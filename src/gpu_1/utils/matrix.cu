#include <fstream>
#include <iomanip>
#include <iostream>

#include "matrix.hh"
#include "utils.hh"

namespace gpu_1::utils
{
    Matrix::Matrix(std::size_t rows, std::size_t cols, value_t value)
        : rows_(rows)
        , cols_(cols)
        , pitch_(0)
        , data_(nullptr)
    {
        hipError_t rc = hipSuccess;
        rc = hipMallocPitch(&this->data_, &this->pitch_, cols * sizeof(value_t), rows);
        if (rc)
        {
            abortError("Fail buffer allocation");
        }
    }

    Matrix::~Matrix()
    {
        hipError_t rc = hipSuccess;
        rc = hipFree(this->data_);
        if (rc)
        {
            abortError("Unable to free memory");
        }
    }

    void Matrix::sub_matrix(std::size_t starting_row,
                            std::size_t starting_col,
                            std::size_t row_count,
                            std::size_t col_count,
                            matrix_device_t& result) const
    {
        sub_matrix_cuda<<<1, 1>>>(
            this->data_, this->pitch_, starting_row, starting_col, row_count, col_count, result.data_, result.pitch_);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
        {
            abortError("Computation Error");
        }
    }

    void Matrix::matrix_transpose(matrix_device_t& result) const
    {
        matrix_transpose_cuda<<<1, 1>>>(
            this->data_, this->pitch_, this->rows_, this->cols_, result.data_, result.pitch_);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
        {
            abortError("Computation Error");
        }
    }

    double Matrix::matrix_norm_2() const
    {
        double *norm_device;
        hipError_t rc = hipSuccess;
        rc = hipMalloc(&norm_device, sizeof(double));
        if (rc)
        {
            abortError("Fail buffer allocation");
        }

        matrix_norm_2_cuda<<<1, 1>>>(this->data_, this->pitch_, this->rows_, this->cols_, norm_device);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
        {
            abortError("Computation Error");
        }

        double norm_host;
        rc = hipMemcpy(&norm_host, norm_device, sizeof(double), hipMemcpyDeviceToHost);
        if (rc)
        {
            abortError("Fail buffer copy");
        }

        rc = hipFree(norm_device);
        if (rc)
        {
            abortError("Fail buffer free");
        }

        return norm_host;
    }

    void Matrix::matrix_subtract_vector(const matrix_device_t& vector, matrix_device_t& result) const
    {
        matrix_subtract_vector_cuda<<<1, 1>>>(this->data_,
                                              this->pitch_,
                                              this->rows_,
                                              this->cols_,
                                              vector.data_,
                                              vector.pitch_,
                                              result.data_,
                                              result.pitch_);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
        {
            abortError("Computation Error");
        }
    }

    void Matrix::matrix_add_vector(const matrix_device_t& vector, matrix_device_t& result) const
    {
        matrix_add_vector_cuda<<<1, 1>>>(this->data_,
                                         this->pitch_,
                                         this->rows_,
                                         this->cols_,
                                         vector.data_,
                                         vector.pitch_,
                                         result.data_,
                                         result.pitch_);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
        {
            abortError("Computation Error");
        }
    }

    void Matrix::matrix_centroid(matrix_device_t& result) const
    {
        matrix_centroid_cuda<<<1, 1>>>(
            this->data_, this->pitch_, this->rows_, this->cols_, result.data_, result.pitch_);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
        {
            abortError("Computation Error");
        }
    }

    void Matrix::multiply_by_scalar(double val, matrix_device_t& result) const
    {
        multiply_by_scalar_cuda<<<1, 1>>>(
            this->data_, this->pitch_, this->rows_, this->cols_, val, result.data_, result.pitch_);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
        {
            abortError("Computation Error");
        }
    }

    double Matrix::matrix_diag_sum() const
    {
        double *sum_device;
        hipError_t rc = hipSuccess;
        rc = hipMalloc(&sum_device, sizeof(double));
        if (rc)
        {
            abortError("Fail buffer allocation");
        }

        matrix_diag_sum_cuda<<<1, 1>>>(this->data_, this->pitch_, this->rows_, sum_device);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
        {
            abortError("Computation Error");
        }

        double sum_host;
        rc = hipMemcpy(&sum_host, sum_device, sizeof(double), hipMemcpyDeviceToHost);
        if (rc)
        {
            abortError("Fail buffer copy");
        }

        rc = hipFree(sum_device);
        if (rc)
        {
            abortError("Fail buffer free");
        }

        return sum_host;
    }

    void Matrix::set_val(std::size_t row, std::size_t col, value_t val)
    {
        set_val_cuda<<<1, 1>>>(this->data_, this->pitch_, row, col, val);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
        {
            abortError("Computation Error");
        }
    }

    void Matrix::set_val_ptr(std::size_t row, std::size_t col, value_t* val)
    {
        set_val_ptr_cuda<<<1, 1>>>(this->data_, this->pitch_, row, col, val);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
        {
            abortError("Computation Error");
        }
    }

    value_t Matrix::get_val(std::size_t row, std::size_t col) const
    {
        value_t *val_device;
        hipError_t rc = hipSuccess;
        rc = hipMalloc(&val_device, sizeof(value_t));
        if (rc)
        {
            abortError("Fail buffer allocation");
        }

        get_val_cuda<<<1, 1>>>(this->data_, this->pitch_, row, col, val_device);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
        {
            abortError("Computation Error");
        }

        double val_host;
        rc = hipMemcpy(&val_host, val_device, sizeof(value_t), hipMemcpyDeviceToHost);
        if (rc)
        {
            abortError("Fail buffer copy");
        }

        rc = hipFree(val_device);
        if (rc)
        {
            abortError("Fail buffer free");
        }

        return val_host;
    }

    void Matrix::print_matrix() const
    {
        std::cout << "rows: " << this->rows_ << " cols: " << this->cols_ << std::endl;
        print_matrix_cuda<<<1, 1>>>(this->data_, this->pitch_, this->rows_, this->cols_);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
        {
            abortError("Computation Error");
        }
    }
} // namespace utils
#include "hip/hip_runtime.h"
#include "matrix.hh"

#include <fstream>
#include <iomanip>
#include <iostream>

#include "lib-matrix.hh"

namespace utils
{
    Matrix::Matrix(std::size_t rows, std::size_t cols, value_t value)
        : rows_(rows)
        , cols_(cols)
    {
        hipError_t rc = hipSuccess;
        std::size_t pitch;

        rc = hipMallocPitch(this->data_, &pitch, cols, rows);
        if (rc)
        {
            abortError("Fail buffer allocation");
        }
    }

    Matrix::~Matrix()
    {
        hipError_t rc = hipSuccess;
        rc = hipFree(this->data_);
        if (rc)
        {
            abortError("Unable to free memory");
        }
    }

    void Matrix::sub_matrix(std::size_t starting_row,
                            std::size_t starting_col,
                            std::size_t row_count,
                            std::size_t col_count,
                            matrix_device_t& result) const
    {
        sub_matrix_cuda<<<1, 1>>>(this, starting_row, starting_col, row_count, col_count, result);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
        {
            abortError("Computation Error");
        }
    }

    void Matrix::matrix_transpose(matrix_device_t& result) const
    {
        matrix_transpose_cuda<<<1, 1>>>(this, result);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
        {
            abortError("Computation Error");
        }
    }

    void Matrix::matrix_norm_2(double& norm) const
    {
        double sum = 0.0;

        for (std::size_t row = 0; row < this->rows_; row++)
        {
            for (std::size_t col = 0; col < this->cols_; col++)
            {
                sum += pow(this->data_[row][col], 2);
            }
        }

        norm = sqrt(sum);
    }

    void Matrix::matrix_subtract_vector(const matrix_device_t& vector, matrix_device_t& result) const
    {
        matrix_subtract_vector_cuda<<<1, 1>>>(this, vector, result);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
        {
            abortError("Computation Error");
        }
    }

    void Matrix::matrix_add_vector(const matrix_device_t& vector, matrix_device_t& result) const
    {
        matrix_add_vector_cuda<<<1, 1>>>(this, vector, result);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
        {
            abortError("Computation Error");
        }
    }

    void Matrix::matrix_centroid(matrix_device_t& result) const
    {
        std::size_t row_count = this->rows_;
        std::size_t col_count = this->cols_;

        for (std::size_t row = 0; row < row_count; row++)
        {
            for (std::size_t col = 0; col < col_count; col++)
            {
                result.data_[0][col] += this->data_[row][col];
            }
        }

        result.data_[0][0] /= row_count;
        result.data_[0][1] /= row_count;
        result.data_[0][2] /= row_count;
    }

    void Matrix::multiply_by_scalar(double val, matrix_device_t& result) const
    {
        multiply_by_scalar_cuda<<<1, 1>>>(this, val, result);
        hipDeviceSynchronize();
        if (hipPeekAtLastError())
        {
            abortError("Computation Error");
        }
    }

    void Matrix::copy_line(const vector_device_t& line, std::size_t row)
    {
        hipMemcpy(this->data_[row], line, this->cols_, hipMemcpyDeviceToDevice);
    }

    void Matrix::copy_line(const parser::vector_host_t& line, std::size_t row)
    {
        // FIXME PAS OUF
        value_t *line_ptr = static_cast<vector_device_t>(malloc(sizeof(value_t) * this->cols_));

        for (std::size_t col = 0; col < this->cols_; col++)
        {
            line_ptr[col] = line[col];
        }

        hipMemcpy(this->data_[row], line, this->cols_, hipMemcpyHostToDevice);

        free(line_ptr);
    }

    void Matrix::copy_data(const value_t& data, std::size_t row, std::size_t col)
    {
        hipMemcpy(this->data_[row][col], &data, sizeof(value_t), hipMemcpyHostToDevice);
    }
} // namespace utils
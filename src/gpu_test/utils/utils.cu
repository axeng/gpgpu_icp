#include "hip/hip_runtime.h"
#include "utils.hh"

namespace utils
{
    __device__ double compute_distance(const vector_t& p, const vector_t& q)
    {
        double X1 = p[0];
        double Y1 = p[1];
        double Z1 = p[2];
        double X2 = q[0];
        double Y2 = q[1];
        double Z2 = q[2];

        return sqrt(pow(X2 - X1, 2) + pow(Y2 - Y1, 2) + pow(Z2 - Z1, 2) * 1.0);
    }

    __global__ void get_nearest_neighbors(matrix_t& P, matrix_t& Q, matrix_t& res, int P_rows, int Q_rows)
    {
        /**
        GPU
        */
        int i = threadIdx.x;
        if (i >= P_rows)
            return;

        float min_dist = MAXFLOAT;
        vector_t chosen;
        for (int ind = 0; ind < Q_rows; ind++)
        {
            auto q_point = Q->data_[ind];
            auto dist = compute_distance(P->data_[i], q_point);
            if (dist < min_dist)
            {
                min_dist = dist;
                chosen = q_point;
            }
        }
        res.set_data(i, chosen);



        /**
        CPU
        */
        /*
        for (const auto& p_point : P)
        {
            float min_dist = MAXFLOAT;

            vector_t chosen;
            for (const auto& q_point : Q)
            {
                auto dist = compute_distance(p_point, q_point);
                if (dist < min_dist)
                {
                    min_dist = dist;
                    chosen = q_point;
                }
            }
            res.emplace_back(chosen);
        }
        */
    }

    void string_split(std::string str, const std::string& delimiter, std::vector<std::string>& words)
    {
        std::size_t position = 0;
        std::string word;

        while ((position = str.find(delimiter)) != std::string::npos)
        {
            word = str.substr(0, position);
            words.push_back(word);
            str.erase(0, position + delimiter.length());
        }

        word = str.substr(0, position);
        words.push_back(word);
    }
} // namespace utils
